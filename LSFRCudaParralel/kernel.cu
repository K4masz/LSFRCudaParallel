#include "hip/hip_runtime.h"

#include <iostream>
#include <chrono>

//DEKLARACAJE FUNKCJI
void initMatrix();
void fillMatrix();
void showMatrix();
void initRegisterState();
void initUsedRegisterValues();
void startCalculations();
void calculateCycle();
void addToUsedRegisterValues();
void calculateRegisterState();
bool calculateCell(bool* matrixRow);
bool isAlreadyUsedState();
bool isEveryRegisterStateUsed();
int parseRegisterState();
void convertToRegisterState(int decimalNumber);

using namespace std;

bool** conversionMatrix;
bool* currentRegisterState;
bool* usedRegisterValues;
int flipFlopCount = 0;
bool maximalCycle = false; //2n-1

long time_start, time_stop;

//FLAGI

bool isEveryRegisterStateUsed()
{
	bool result = true;

	int n = pow(2, flipFlopCount) - 1;

	for (int i = 1; i <= n; i++)
		if (usedRegisterValues[i] == false)
		{
			result = false;
			break;
		}

	return result;
}

bool isAlreadyUsedState()
{
	return usedRegisterValues[parseRegisterState()];
}

//INIT

void initMatrix()
{
	cout << "Podaj wielko�� macierzy: ";
	cin >> flipFlopCount;
	cout << endl << endl;

	hipMallocManaged(&conversionMatrix, pow(flipFlopCount, 2) * sizeof(bool));

	//INITIALIZING MATRIX
	conversionMatrix = new bool *[flipFlopCount];
	for (int i = 0; i < flipFlopCount; i++)
	{
		conversionMatrix[i] = new bool[flipFlopCount];
	}
	//INITIALIZING MATRIX - END
}

void initUsedRegisterValues()
{
	int n = pow(2, flipFlopCount); // na 3 bitach jest 8 stan�w - nie 7
	usedRegisterValues = new bool[n]{false};
	usedRegisterValues[0] = 1;
}

void initRegisterState()
{
	hipMallocManaged(&currentRegisterState, flipFlopCount * sizeof(bool));
	currentRegisterState = new bool[flipFlopCount];
	//currentRegisterState[0] = 1;
	for (int i = 0; i < flipFlopCount; i++)
	{
		currentRegisterState[i] = 0;
	}
}

void printRegisterState()
{
	for (int i = flipFlopCount-1; i >= 0; i--)
	{
		cout << currentRegisterState[i] ;
	}
	cout<< endl;
}

//MATRIX

void fillMatrix()
{
	for (int i = 0; i < flipFlopCount; i++)
	{
		cout << "Podaj " << i + 1 << " wiersz: ";

		for (int j = 0; j < flipFlopCount; j++)
		{
			cin >> conversionMatrix[i][j];
		}
		cout << endl;
	}
}

void showMatrix()
{
	cout << endl;

	for (int i = 0; i < flipFlopCount; i++)
	{
		for (int j = 0; j < flipFlopCount; j++)
		{
			cout << conversionMatrix[i][j] << " ";
		}

		cout << endl;
	}
}

__global__ void calculateCellParallel(bool* currentRS, bool* nextRS, bool** matrix, int n)
{
	int result = 0;
	
	for (int i = 0; i < n; i++)
	{
		result = result + (matrix[threadIdx.x][i] * currentRS[i]);
		
	}
	
	 nextRS[threadIdx.x] = (result % 2);
}

bool calculateCell(bool* matrixRow)
{
	int result = 0;

	for (int i = 0; i < flipFlopCount; i++)
	{
		result = result + (matrixRow[i] * currentRegisterState[i]);
	}

	return (result % 2);
}

void calculateRegisterState()
{
	bool* nextRegister = new bool[flipFlopCount];

	for (int i = 0; i < flipFlopCount; i++)
	{
		nextRegister[i] = calculateCell(conversionMatrix[i]);
	}

	currentRegisterState = nextRegister;
}

void calculateRegisterStateParallel()
{
	cout << "START ";

	bool* nextRegisterState;

	hipMallocManaged(&nextRegisterState, flipFlopCount * sizeof(bool));
	// cudaMallocManaged(&conversionMatrix, pow(flipFlopCount, 2) * sizeof(bool));
	// cudaMallocManaged(&currentRegisterState, flipFlopCount * sizeof(bool));

	nextRegisterState = new bool[flipFlopCount]{false};


	cout << "GPU";
	calculateCellParallel <<<1, flipFlopCount>>>(currentRegisterState, nextRegisterState, conversionMatrix,
	                                             flipFlopCount);


	hipDeviceSynchronize();
	cout << " STOP ";

	*currentRegisterState = nextRegisterState;

	hipFree(nextRegisterState);
	printRegisterState();
}

//LOGIKA


void calculateCycle()
{
	//przemno�y� macierz przez rejestr - mamy rejestr wynikowy
	int statesNumber = 0;
	do
	{
		statesNumber++;
		addToUsedRegisterValues();
		calculateRegisterStateParallel();
	}
	while ((parseRegisterState() != 0) && !isAlreadyUsedState());
	//sprawdzi� czy rejestr przedstawia zero je�li jest lub czy jest w u�ytych (parsowane na liczbe) - jesli tak koniec
	cout << "liczba stan�w w cyklu: " << statesNumber << endl;
	if (statesNumber == pow(2, flipFlopCount) - 1)
	{
		maximalCycle = true;
	}
}

int parseRegisterState()
{
	//metoda parsuj�ca
	int result = 0;
	for (int i = 0; i < flipFlopCount; i++)
		if (currentRegisterState[i])
			result += pow(2, i);

	return result;
}

void addToUsedRegisterValues()
{
	usedRegisterValues[parseRegisterState()] = true;
}

void findFirstAvailableState()
{
	int n = pow(flipFlopCount, 2) - 1;

	for (int i = 1; i <= n; i++)
	{
		if (!usedRegisterValues[i])
		{
			convertToRegisterState(i);
			break;
		}
	}
}

void convertToRegisterState(int decimalNumber)
{
	int i = 0;
	do
	{
		currentRegisterState[i] = decimalNumber % 2;
		decimalNumber /= 2;
		i++;
	}
	while (decimalNumber > 0);
}

void startCalculations()
{
	time_start = std::chrono::system_clock::now().time_since_epoch().count();


	//Pocz�tek wszystkich oblicze�
	int cyclesNumber = 0;
	//pocz�tek liczenia jednego cyklu
	do
	{
		//obecny rejestr idzie do u�ytych rejestr�w
		findFirstAvailableState();
		calculateCycle();
		//koniec p�tli
		cyclesNumber++;
	}
	while (!isEveryRegisterStateUsed());

	cout << endl << "Finalna liczba cykli: " << cyclesNumber << endl;

	if (maximalCycle)
	{
		cout << endl << "generuje cykl maksymalny" << endl;;
	}
	time_stop = (std::chrono::system_clock::now().time_since_epoch().count());
	cout << "Algorytm trwa�: " << time_stop - time_start << "ms" << endl;
}


//MAIN

int main()
{
	
	initMatrix();
	fillMatrix();
	showMatrix();

	initRegisterState();
	initUsedRegisterValues();

	startCalculations();

	getchar();
	return 0;
}
